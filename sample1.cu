#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <functional>
#include <iostream>
#include <random>
#include <vector>
#include <cstdarg>

#define DEFAULT_SPACE_DIM 30

// # define running mode before the 'core'.

#include "calib/mode/host_device.hpp"
//#include "calib/mode/host_only.hpp"

// # choose default metric space.

//#include "calib/geometries/euclidian.hpp"

#include "calib/core.hpp"

using namespace calib;

multivector random_generate_multivector ()
{
    std::srand (time (0));

    multivector m1;

    // # random number of elements .

    int dim = DEFAULT_SPACE_DIM;

    for (int i = 0; i < dim; i++)
    {
        // # random degree of current element.

        int degree = (std::rand() % (dim))+1;

        std::vector <int> indices;

        for (int j = 0; j < degree; j++)
        {
            int random = (std::rand() % (DEFAULT_SPACE_DIM))+1;
            indices. push_back (random != 0 ? random : 1);
        }

        basis base (indices);
        base. magnitude = std::rand () % 100;
        base. orientation = std::copysign (1, (std::rand() % 2) - 1);

        m1. add_elem (base);
    }

    cannonical_reordering (m1);

    return m1;
}

int main (int argv, char * argc)
{
    auto e1 = basis ({2,4,3}) * -3;
    auto e2 = basis ({2}) * 2;
    auto e3 = basis ({3,2,1}) * -7;

    //std::cout << "e1: " << e1 << std::endl;
    //std::cout << "e2: " << e2 << std::endl;

    multivector m1 = e1 + e2;
    multivector m2 = e2 + e3;

    auto m3 = _outer_prd_ (m1, m2);
    auto m4 = _regr_prd_ (m1, m2);
    float inner_prd = _inner_prd_ (m1, m2, euclidian_metric ());

    //std::cout << "m1: " << m1 << std::endl;
    //std::cout << "m2: " << m2 << std::endl;

    //std::cout << "m1 . m2: " << inner_prd << std::endl;

#ifdef CALIB_MODE_HOST_DEVICE
    // # init constant memory

    float * fact;
    fact = (float *) std::malloc (CALIB_FACTORIAL_TABLE_SIZE * sizeof (float));

    fact [0] = 1;
    for (int i = 1; i < CALIB_FACTORIAL_TABLE_SIZE; i++)
    {
        fact [i] = i * fact [i-1];
    }

    hipMemcpyToSymbol(HIP_SYMBOL(fact_), &fact, CALIB_FACTORIAL_TABLE_SIZE * sizeof(float));
#endif

    // # rules of the experiment.

    int n = 1000;
    double mean_time = 0;


    for (int i = 0; i < n; i++)
    {
        // # random generation of

        multivector m1 = random_generate_multivector ();
        multivector m2 = random_generate_multivector ();

        #ifdef CALIB_MODE_HOST_ONLY
            std::clock_t c_start = std::clock();
        #endif

        #ifdef CALIB_MODE_HOST_DEVICE
            hipEvent_t start, stop;

            hipEventCreate (&start);
            hipEventCreate (&stop);

            hipEventRecord (start, 0);
        #endif

        //m1 ^ m2;
        //_regr_prd_ (m1, m2);
        _inner_prd_ (m1, m2, euclidian_metric ());

        #ifdef CALIB_MODE_HOST_ONLY
            std::clock_t c_end   = std::clock();
            mean_time += 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
        #endif

        #ifdef CALIB_MODE_HOST_DEVICE
            hipEventRecord (stop, 0);

            hipEventSynchronize (stop);

            float time_elapsed_ms;
            hipEventElapsedTime (&time_elapsed_ms, start, stop);
            mean_time += time_elapsed_ms;
            
            hipEventDestroy (start);
            hipEventDestroy (stop);
        #endif

        std::cout << ".";
    }

#ifdef CALIB_MODE_HOST_DEVICE
    hipFree (&fact_);
#endif

    std::cout << std::endl;

    std::cout << "Experiment with " << DEFAULT_SPACE_DIM << " dimensions." << std::endl;
    std::cout << "> running after " << n << " times." << std::endl;
    std::cout << "> mean time: " << mean_time / n << " ms." << std::endl;
}